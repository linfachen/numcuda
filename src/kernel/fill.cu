#include "hip/hip_runtime.h"
#include <stdint.h>
#include <iostream>
#include "util.h" 
#include "hip/hip_fp16.h"

template<typename T>
__global__ void fill(T *data,T value,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<size){
        *(data+i) = value;
    } 
}


void fill_value(char *data,double value,int size,Dtype dtype,int thread_size=32)
{
    int block_size = std::ceil(size*1.0/thread_size);
    switch(dtype){
        case Dtype::float16:
            fill<<<block_size,thread_size>>>((__half*)data,__float2half(static_cast<float>(value)),size);
            break;
        case Dtype::float32:
            fill<<<block_size,thread_size>>>((float*)data,static_cast<float>(value),size);
            break;
        case Dtype::float64: 
            fill<<<block_size,thread_size>>>((double*)data,value,size);
            break;
        default:
            std::cerr<<"use wrong type"<<std::endl;           
    }
}


void fill_value(char *data,long long value,int size,Dtype dtype,int thread_size=32)
{
    int block_size = std::ceil(size*1.0/thread_size);
    switch(dtype){
        case Dtype::int8:
        fill<<<block_size,thread_size>>>((int8_t*)data,static_cast<int8_t>(value),size);
        break;
        case Dtype::uint8:
        fill<<<block_size,thread_size>>>((uint8_t*)data,static_cast<uint8_t>(value),size);
        break;
        case Dtype::int16:
        fill<<<block_size,thread_size>>>((int16_t*)data,static_cast<int16_t>(value),size);
        break;
        case Dtype::uint16:
        fill<<<block_size,thread_size>>>((uint16_t*)data,static_cast<uint16_t>(value),size);
        break;
        case Dtype::int32:
        fill<<<block_size,thread_size>>>((int32_t*)data,static_cast<int32_t>(value),size);
        break;
        case Dtype::uint32:
        fill<<<block_size,thread_size>>>((uint32_t*)data,static_cast<uint32_t>(value),size);
        break;
        case Dtype::int64:
        fill<<<block_size,thread_size>>>((int64_t*)data,static_cast<int64_t>(value),size);
        break;
        case Dtype::uint64:
        fill<<<block_size,thread_size>>>((uint64_t*)data,static_cast<uint64_t>(value),size);
        break;
        default:
            std::cerr<<"use wrong type"<<std::endl;   
    }
}
#include "hip/hip_runtime.h"
#include <stdint.h>
#include <iostream>
#include "util.h" 
#include "hip/hip_fp16.h"

//add kernel
template<typename T>
__global__ void elt_add(T *a,T *b,T *c,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<size){
        *(c+i) = *(a+i) + *(b+i);
    } 
}

template<>
__global__ void elt_add(__half *a,__half *b,__half *c,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<size){
        *(c+i) = __hadd(*(a+i), *(b+i));
    } 
}

//sub kernel
template<typename T>
__global__ void elt_sub(T *a,T *b,T *c,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<size){
        *(c+i) = *(a+i) - *(b+i);
    }
}

template<>
__global__ void elt_sub(__half *a,__half *b,__half *c,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<size){
        *(c+i) = __hsub(*(a+i), *(b+i));
    }
}


//mul kernel
template<typename T>
__global__ void elt_mul(T *a,T *b,T *c,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<size){
        *(c+i) = *(a+i) * *(b+i);
    }
}

template<>
__global__ void elt_mul(__half *a,__half *b,__half *c,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<size){
        *(c+i) = __hmul(*(a+i), *(b+i));
    }
}

//div kernel
template<typename T>
__global__ void elt_div(T *a,T *b,T *c,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<size){
        *(c+i) = *(a+i) / *(b+i);
    }
}

template<>
__global__ void elt_div(__half *a,__half *b,__half *c,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<size){
        *(c+i) = __hdiv(*(a+i), *(b+i));
    }
}



#define REGSITER_BINARY_OP(name) \
void name##_op(char *a,char *b,char *c,int size,Dtype dtype,int thread_size=32)\
{\
    int block_size = std::ceil(size*1.0/(elem_size(dtype)*thread_size));\
    switch(dtype){\
        case Dtype::float16:\
            name<<<block_size,thread_size>>>((__half*)a,(__half*)b,(__half*)c,size);\
            break;\
        case Dtype::float32:\
            name<<<block_size,thread_size>>>((float*)a,(float*)b,(float*)c,size);\
            break;\
        case Dtype::float64:\
            name<<<block_size,thread_size>>>((double*)a,(double*)b,(double*)c,size);\
            break;\
        case Dtype::int8:\
            name<<<block_size,thread_size>>>((int8_t*)a,(int8_t*)b,(int8_t*)c,size);\
            break;\
        case Dtype::uint8:\
            name<<<block_size,thread_size>>>((uint8_t*)a,(uint8_t*)b,(uint8_t*)c,size);\
            break;\
        case Dtype::int16:\
            name<<<block_size,thread_size>>>((int16_t*)a,(int16_t*)b,(int16_t*)c,size);\
            break;\
        case Dtype::uint16:\
            name<<<block_size,thread_size>>>((uint16_t*)a,(uint16_t*)b,(uint16_t*)c,size);\
            break;\
        case Dtype::int32:\
            name<<<block_size,thread_size>>>((int32_t*)a,(int32_t*)b,(int32_t*)c,size);\
            break;\
        case Dtype::uint32:\
            name<<<block_size,thread_size>>>((uint32_t*)a,(uint32_t*)b,(uint32_t*)c,size);\
            break;\
        case Dtype::int64:\
            name<<<block_size,thread_size>>>((int64_t*)a,(int64_t*)b,(int64_t*)c,size);\
            break;\
        case Dtype::uint64:\
            name<<<block_size,thread_size>>>((uint64_t*)a,(uint64_t*)b,(uint64_t*)c,size);\
            break;\
        default:\
            std::cerr<<"use wrong type"<<std::endl;\
    }\
}\


//regsiter op
REGSITER_BINARY_OP(elt_add)
REGSITER_BINARY_OP(elt_sub)
REGSITER_BINARY_OP(elt_mul)
REGSITER_BINARY_OP(elt_div)
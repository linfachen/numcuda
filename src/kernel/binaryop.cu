#include "hip/hip_runtime.h"
#include <stdint.h>
#include <iostream>
#include "util.h" 
#include "hip/hip_fp16.h"

template<typename T>
__global__ void elt_add(T *a,T *b,T *c,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<size){
        *(c+i) = *(a+i) + *(b+i);
    } 
}

template<>
__global__ void elt_add(__half *a,__half *b,__half *c,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<size){
        *(c+i) = __hadd(*(a+i), *(b+i));
    } 
}


void elt_add_op(char *a,char *b,char *c,int size,Dtype dtype,int thread_size=32)
{
    int block_size = std::ceil(size*1.0/thread_size);
    switch(dtype){
        case Dtype::float16:
            elt_add<<<block_size,thread_size>>>((__half*)a,(__half*)b,(__half*)c,size);
            break;
        case Dtype::float32:
            elt_add<<<block_size,thread_size>>>((float*)a,(float*)b,(float*)c,size);
            break;
        case Dtype::float64: 
            elt_add<<<block_size,thread_size>>>((double*)a,(double*)b,(double*)c,size);
            break;
        case Dtype::int8:
            elt_add<<<block_size,thread_size>>>((int8_t*)a,(int8_t*)b,(int8_t*)c,size);
            break;
        case Dtype::uint8:
            elt_add<<<block_size,thread_size>>>((uint8_t*)a,(uint8_t*)b,(uint8_t*)c,size);
            break;
        case Dtype::int16:
            elt_add<<<block_size,thread_size>>>((int16_t*)a,(int16_t*)b,(int16_t*)c,size);
            break;
        case Dtype::uint16:
            elt_add<<<block_size,thread_size>>>((uint16_t*)a,(uint16_t*)b,(uint16_t*)c,size);
            break;
        case Dtype::int32:
            elt_add<<<block_size,thread_size>>>((int32_t*)a,(int32_t*)b,(int32_t*)c,size);
            break;
        case Dtype::uint32:
            elt_add<<<block_size,thread_size>>>((uint32_t*)a,(uint32_t*)b,(uint32_t*)c,size);
            break;
        case Dtype::int64:
            elt_add<<<block_size,thread_size>>>((int64_t*)a,(int64_t*)b,(int64_t*)c,size);
            break;
        case Dtype::uint64:
            elt_add<<<block_size,thread_size>>>((uint64_t*)a,(uint64_t*)b,(uint64_t*)c,size);
            break;
        default:
            std::cerr<<"use wrong type"<<std::endl;                
    }
}


